#include "hip/hip_runtime.h"
#include "SystemDefines.cuh"

void handleError(hipError_t cu) {
	if (cu != hipSuccess) {
		printf("%s\n", hipGetErrorString(cu));
		system("pause");
		exit(0);
	}
}

Scalar RectangularBox::Volume() {
	return Length*Width*Height;
}

__device__ __host__ Scalar Scalar3::norm() {
	return sqrt(x*x + y*y + z*z);
}

__device__ __host__  Scalar Scalar3::norm2() {
	return x*x + y*y + z*z;
}